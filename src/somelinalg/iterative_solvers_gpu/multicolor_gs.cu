#include "hip/hip_runtime.h"
// fused_multicolor_gs.cu
/*
nvcc -shared -Xcompiler "/MD" ^
  -gencode arch=compute_52,code=sm_52 ^
  -gencode arch=compute_52,code=compute_52 ^
  multicolor_gs.cu -o gsgpu.dll


*/

// cuda/multicolor_gs.cu
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cstdio>

using namespace cooperative_groups;

extern "C" {

    // simple per-color kernel
__global__ void per_color_kernel(
    int n, int num_diags, const int *offsets,
    const float *A_diags, const float *b,
    float *x, int color, int color_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        if ((i % color_count) != color) continue;
        float diag = 0.0f;
        float sum = 0.0f;
        for (int k = 0; k < num_diags; ++k) {
            int off = offsets[k];
            int j = i + off;
            float a_ij = A_diags[k * n + i];
            if (off == 0) diag = a_ij;
            else if (off < 0) {
                if (j >= 0 && j < n) sum += a_ij * x[j];
            }
        }
        if (diag != 0.0f) x[i] = (b[i] - sum) / diag;
    }
}


// fused kernel: does forward pass; if symmetric==1 it does backward pass as well.
// A_diags layout: column-major (diag k at A_diags + k*n, element for row i is A_diags[k*n + i])
// offsets: offsets[k] corresponds to diagonal k; can be negative (lower), 0 (main), positive (upper).
__global__ void fused_multicolor_gs_kernel(
    int n,
    int num_diags,
    const int *offsets,          // length num_diags
    const float *A_diags,        // length num_diags * n, column-major by diag
    const float *b,
    float *x,
    int color_count,
    int symmetric)
{
    grid_group grid = this_grid();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // forward-sweep compute (lower part)
    auto compute_forward = [&](int idx) -> float {
        float diag = 0.0f;
        float sum  = 0.0f;
        for (int k = 0; k < num_diags; ++k) {
            int off = offsets[k];
            int j = idx + off;
            float a_ij = A_diags[k * n + idx];
            if (off == 0) {
                diag = a_ij;
            } else if (off < 0) {
                if (j >= 0 && j < n)
                    sum += a_ij * x[j];
            }
        }
        if (diag == 0.0f) return x[idx];
        return (b[idx] - sum) / diag;
    };

    // backward-sweep compute (upper part)
    auto compute_backward = [&](int idx) -> float {
        float diag = 0.0f;
        float sum  = 0.0f;
        for (int k = 0; k < num_diags; ++k) {
            int off = offsets[k];
            int j = idx + off;
            float a_ij = A_diags[k * n + idx];
            if (off == 0) {
                diag = a_ij;
            } else if (off > 0) { // use upper diags now
                if (j >= 0 && j < n)
                    sum += a_ij * x[j];
            }
        }
        if (diag == 0.0f) return x[idx];
        return (b[idx] - sum) / diag;
    };

    // Forward pass (lower)
    for (int color = 0; color < color_count; ++color) {
        for (int idx = tid; idx < n; idx += stride) {
            if ((idx % color_count) != color) continue;
            x[idx] = compute_forward(idx);
        }
        grid.sync();
    }

    // Backward pass (upper)
    if (symmetric) {
        for (int color = color_count - 1; color >= 0; --color) {
            for (int idx = tid; idx < n; idx += stride) {
                if ((idx % color_count) != color) continue;
                x[idx] = compute_backward(idx);
            }
            grid.sync();
        }
    }
}

// host-side launcher: tries cooperative launch and falls back to per-color launches
__declspec(dllexport)
int launch_multicolor_gs_fused(
    int n,
    int num_diags,
    const int *offsets_dev,
    const float *A_diags_dev,
    const float *b_dev,
    float *x_dev,
    int color_count,
    int symmetric)
{
    int device;
    hipError_t cerr = hipGetDevice(&device);
    if (cerr != hipSuccess) {
        printf("[CUDA] hipGetDevice failed: %s\n", hipGetErrorString(cerr));
        return -10;
    }

    int attr = 0;
    hipDeviceGetAttribute(&attr, hipDeviceAttributeCooperativeLaunch, device);
    int cooperative = attr;

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

  // printf("[CUDA] launch_multicolor_gs_fused: n=%d, num_diags=%d, blocks=%d threads=%d coop=%d\n",
      //     n, num_diags, blocks, threads, cooperative);
   // fflush(stdout);

    if (cooperative) {
        void *kernelArgs[] = {
            (void *)&n,
            (void *)&num_diags,
            (void *)&offsets_dev,
            (void *)&A_diags_dev,
            (void *)&b_dev,
            (void *)&x_dev,
            (void *)&color_count,
            (void *)&symmetric};
        cerr = hipLaunchCooperativeKernel(
            (void*)fused_multicolor_gs_kernel,
            blocks, threads,
            kernelArgs);
        if (cerr != hipSuccess) {
            printf("[CUDA] cooperative launch failed: %s\n", hipGetErrorString(cerr));
        } else {
            hipDeviceSynchronize();
            printf("[CUDA] cooperative fused kernel finished\n");
            return 0;
        }
    }

    // Fallback: per-color launches (portable)
  //  printf("[CUDA] Falling back to per-color launches (color_count=%d)\n", color_count);
    for (int c = 0; c < color_count; ++c) {
        per_color_kernel<<<blocks, threads>>>(
            n, num_diags, offsets_dev, A_diags_dev, b_dev, x_dev, c, color_count);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("[CUDA] per-color launch error color %d: %s\n", c, hipGetErrorString(err));
            return 2 + c;
        }
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("[CUDA] per-color sync failed color %d: %s\n", c, hipGetErrorString(err));
            return 100 + c;
        }
    }
        // if symmetric requested, do backward pass also
    if (symmetric) {
        for (int c = color_count - 1; c >= 0; --c) {
            per_color_kernel<<<blocks, threads>>>(
                n, num_diags, offsets_dev, A_diags_dev, b_dev, x_dev, c, color_count
            );
            
                  hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("[CUDA] per-color launch error color %d: %s\n", c, hipGetErrorString(err));
            return 2 + c;
        }
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("[CUDA] per-color sync failed color %d: %s\n", c, hipGetErrorString(err));
            return 100 + c;
        }

        }
    }
    return 0;
}


// host helper: allocate device buffers, copy data, run kernel, copy back
__declspec(dllexport)
int launch_multicolor_gs_host(
    int n,
    int num_diags,
    const int *offsets_host,
    const float *a_diags_host,
    const float *b_host,
    float *x_host,
    int color_count,
    int symmetric)
{
    size_t bytes_diags = (size_t)num_diags * n * sizeof(float);
    size_t bytes_offs = (size_t)num_diags * sizeof(int);
    size_t bytes_vec = (size_t)n * sizeof(float);

    float *d_diags = nullptr, *d_b = nullptr, *d_x = nullptr;
    int *d_offsets = nullptr;

    hipError_t err;
    err = hipMalloc((void**)&d_diags, bytes_diags); if (err != hipSuccess) return -1;
    err = hipMalloc((void**)&d_b, bytes_vec);       if (err != hipSuccess) { hipFree(d_diags); return -2; }
    err = hipMalloc((void**)&d_x, bytes_vec);       if (err != hipSuccess) { hipFree(d_diags); hipFree(d_b); return -3; }
    err = hipMalloc((void**)&d_offsets, bytes_offs);if (err != hipSuccess) { hipFree(d_diags); hipFree(d_b); hipFree(d_x); return -4; }

    err = hipMemcpy(d_diags, a_diags_host, bytes_diags, hipMemcpyHostToDevice); if (err != hipSuccess) goto cleanup;
    err = hipMemcpy(d_b, b_host, bytes_vec, hipMemcpyHostToDevice);             if (err != hipSuccess) goto cleanup;
    err = hipMemcpy(d_x, x_host, bytes_vec, hipMemcpyHostToDevice);             if (err != hipSuccess) goto cleanup;
    err = hipMemcpy(d_offsets, offsets_host, bytes_offs, hipMemcpyHostToDevice);if (err != hipSuccess) goto cleanup;

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);

    // cooperative launch attempt
    int coop_attr = 0;
    hipDeviceGetAttribute(&coop_attr, hipDeviceAttributeCooperativeLaunch, 0);
    if (coop_attr) {
        void* kernelArgs[] = {
            (void*)&n, (void*)&num_diags, (void*)&d_offsets,
            (void*)&d_diags, (void*)&d_b, (void*)&d_x,
            (void*)&color_count, (void*)&symmetric };
        hipError_t cerr = hipLaunchCooperativeKernel(
            (void*)fused_multicolor_gs_kernel,
            grid.x, block.x, kernelArgs);
        if (cerr == hipSuccess) {
            cerr = hipDeviceSynchronize();
            if (cerr != hipSuccess) { err = cerr; goto cleanup; }
        } else {
            // fallback below
        }
    }

    // fallback per-color launches (forward)
    for (int c = 0; c < color_count; ++c) {
        per_color_kernel<<<grid, block>>>(n, num_diags, d_offsets, d_diags, d_b, d_x, c, color_count);
        err = hipGetLastError(); if (err != hipSuccess) goto cleanup;
        err = hipDeviceSynchronize(); if (err != hipSuccess) goto cleanup;
    }

    // if symmetric requested, do backward pass also
    if (symmetric) {
        for (int c = color_count - 1; c >= 0; --c) {
            per_color_kernel<<<grid, block>>>(n, num_diags, d_offsets, d_diags, d_b, d_x, c, color_count);
            err = hipGetLastError(); if (err != hipSuccess) goto cleanup;
            err = hipDeviceSynchronize(); if (err != hipSuccess) goto cleanup;
        }
    }

    err = hipMemcpy(x_host, d_x, bytes_vec, hipMemcpyDeviceToHost);
    if (err != hipSuccess) goto cleanup;

    hipFree(d_diags); hipFree(d_b); hipFree(d_x); hipFree(d_offsets);
    return 0;

cleanup:
    hipFree(d_diags); hipFree(d_b); hipFree(d_x); hipFree(d_offsets);
    return 10 + (int)err;
}

} // extern "C"


////////////////////////////////TESTS////////////////////////////////////////////////////
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C" {

__global__ void increment_kernel(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] += 1.0f;
        if (idx < 3) {
            printf("[KERNEL] Thread %d: data[%d] = %.1f\n", idx, idx, data[idx]);
        }
    }
}

// Return codes:
//  0 = success
// <0 = hipMalloc or hipMemcpy H2D failed
// >0 = hipMemcpy D2H or kernel sync failed (positive codes)
__declspec(dllexport)
int launch_multicolor_gs_test(float *host_data, int n) {
    printf("[CUDA] launch_multicolor_gs_test called with n=%d, host_data=%p\n", n, host_data);
    fflush(stdout);

    float *dev_data = nullptr;
    size_t bytes = (size_t)n * sizeof(float);
    hipError_t err;

    err = hipMalloc((void**)&dev_data, bytes);
    if (err != hipSuccess) {
        printf("[CUDA] hipMalloc failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
        return -1;
    }

    // Print host values we will copy
    printf("[CUDA] First 3 input values (host): %.6f, %.6f, %.6f\n",
           (n>0?host_data[0]:0.0f), (n>1?host_data[1]:0.0f), (n>2?host_data[2]:0.0f));
    fflush(stdout);

    err = hipMemcpy(dev_data, host_data, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("[CUDA] hipMemcpy H2D failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
        hipFree(dev_data);
        return -2;
    }

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    printf("[CUDA] Launching kernel with blocks=%d, threads=%d\n", blocks, threads);
    fflush(stdout);

    increment_kernel<<<blocks, threads>>>(dev_data, n);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("[CUDA] Kernel launch failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
        hipFree(dev_data);
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("[CUDA] Kernel execution failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
        hipFree(dev_data);
        return 2;
    }

    err = hipMemcpy(host_data, dev_data, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("[CUDA] hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        fflush(stdout);
        hipFree(dev_data);
        return 3;
    }

    printf("[CUDA] First 3 output values (host after D2H): %.6f, %.6f, %.6f\n",
           (n>0?host_data[0]:0.0f), (n>1?host_data[1]:0.0f), (n>2?host_data[2]:0.0f));
    fflush(stdout);

    hipFree(dev_data);
    return 0;
}

__declspec(dllexport)
void get_gpu_info() {
    printf("[CUDA] get_gpu_info called\n");
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("[CUDA] hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return;
    }
    
    printf("[CUDA] Found %d CUDA devices\n", deviceCount);
    
    if (deviceCount == 0) {
        printf("[CUDA] No CUDA devices found!\n");
        return;
    }
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess) {
            printf("[CUDA] hipGetDeviceProperties failed for device %d: %s\n", i, hipGetErrorString(err));
            continue;
        }
        
        printf("[CUDA] Device %d: %s\n", i, prop.name);
        printf("[CUDA] Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("[CUDA] SM count: %d\n", prop.multiProcessorCount);
        printf("[CUDA] Global memory: %.1f MB\n", prop.totalGlobalMem / (1024.0f * 1024.0f));
    }
}

__declspec(dllexport)
void increment_device_data(float *dev_data, int n) {
    printf("[CUDA] increment_device_data called with n=%d, dev_data=%p\n", n, dev_data);
    
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    
    increment_kernel<<<blocks, threads>>>(dev_data, n);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("[CUDA] Kernel launch error: %s\n", hipGetErrorString(err));
        return;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("[CUDA] Sync error: %s\n", hipGetErrorString(err));
        return;
    }
    
    printf("[CUDA] increment_device_data completed successfully\n");
}

} // extern "C"